/*
 * Copyright (c) 2018-2024 curoky(cccuroky@gmail.com).
 *
 * This file is part of minimal-example.
 * See https://github.com/curoky/minimal-example for further info.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(expr)                                                     \
  do {                                                                       \
    hipError_t err = (expr);                                                \
    if (err != hipSuccess) {                                                \
      fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n", err, \
              hipGetErrorString(err));                                      \
      exit(err);                                                             \
    }                                                                        \
  } while (0)

int main() {
  int deviceCount;
  CUDA_CHECK(hipGetDeviceCount(&deviceCount));
  std::cout << "deviceCount: " << deviceCount << std::endl;
  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t devProp;
    CUDA_CHECK(hipGetDeviceProperties(&devProp, i));
    std::cout << "# GPU device " << i << ": " << devProp.name << std::endl;
    std::cout << "-> totalGlobalMem: " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    std::cout << "-> SM count" << devProp.multiProcessorCount << std::endl;
    std::cout << "-> sharedMemPerBlock: " << devProp.sharedMemPerBlock / 1024.0 << " KB"
              << std::endl;
    std::cout << "-> maxThreadsPerBlock: " << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "-> regsPerBlock: " << devProp.regsPerBlock << std::endl;
    std::cout << "-> maxThreadsPerMultiProcessor: " << devProp.maxThreadsPerMultiProcessor
              << std::endl;
    std::cout << "-> maxThreadsPerMultiProcessor: " << devProp.maxThreadsPerMultiProcessor / 32
              << std::endl;
    std::cout << "-> multiProcessorCount: " << devProp.multiProcessorCount << std::endl;
  }
  return 0;
}
